#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include "mat.h"

#include "helpers.cuh"

void computeForwardPass() {

}


int main()
{

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    gpuErrchk(hipDeviceReset());

    return 0;
}