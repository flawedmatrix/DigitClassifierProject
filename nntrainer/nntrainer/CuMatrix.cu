#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "CuMatrix.cuh"

// non-specialized class template
template <class T>
class SharedMem
{
public:
    // Ensure that we won't compile any un-specialized types
    T* getPointer() { exit(1); };
};

// specialization for int
template <>
class SharedMem <int>
{
public:
    __device__ int* getPointer() { extern __shared__ int s_int[]; return s_int; }
};

// specialization for float
template <>
class SharedMem <float>
{
public:
    __device__ float* getPointer() { extern __shared__ float s_float[]; return s_float; }
};

template <class T>
__global__ void matrixAdd(const T *A, const T *B, T *C, const size_t numElements) {
    unsigned int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    unsigned int grid_width = gridDim.x * blockDim.x;
    unsigned int i = i1 * grid_width + i0;
    
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

template <class T>
__global__ void matrixAdd2(const T *A, const T *B, T *C, const size_t numElements) {
    unsigned int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    unsigned int grid_width = gridDim.x * blockDim.x;
    unsigned int i = i1 * grid_width + i0;
    
    if (i < numElements)
    {
        C[i] = A[i] + B[i0];
    }
}

template <class T>
__global__ void matrixHadm(const T *A, const T *B, T *C, const size_t numElements) {
    unsigned int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    unsigned int grid_width = gridDim.x * blockDim.x;
    unsigned int i = i1 * grid_width + i0;
    
    if (i < numElements)
    {
        C[i] = A[i] * B[i];
    }
}

__global__ void matrixApplySigmoid(float *A, const size_t numElements) {
    unsigned int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = blockIdx.y * blockDim.y + threadIdx.y;

    // map the two 2D indices to a single linear, 1D index
    unsigned int grid_width = gridDim.x * blockDim.x;
    unsigned int i = i1 * grid_width + i0;
    
    if (i < numElements)
    {
        float z = A[i];
        float denom = 1 + exp(-z);
        A[i] = 1/denom;
    }
}

template <class T>
__global__ void reduction(const T *A, T *B, const size_t numElements) {
    SharedMem<T> shared;
    T* sdata = shared.getPointer();

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    T x = 0;
    if (i < numElements) {
        x = A[i];
    }
    // each thread loads one element from global to shared mem
    sdata[tid] = x;
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        // wait until all threads in the block have updated their partial sums
        __syncthreads();
    }
    // thread 0 writes the per-block result
    if (tid == 0) B[blockIdx.x] = sdata[0];
}

hipblasHandle_t CuBase::cuHandle = nullptr;

template class CuMatrix<int>;
template class CuMatrix<float>;

void CuBase::initializeHandle() {
    // Create a handle for CUBLAS
    hipblasCreate(&cuHandle);
}

void CuBase::closeHandle() {
    // Destroy the handle
    hipblasDestroy(cuHandle);
}

template <class T>
CuMatrix<T>::CuMatrix(int rows, int cols):
    d0(rows), d1(cols), gpuData(0)
{
}

template <class T>
CuMatrix<T>::CuMatrix(CuMatrix<T> &m) {
    d0 = m.d0;
    d1 = m.d1;
    gpuErrchk(hipMalloc((void**)&gpuData, d0 * d1 * sizeof(T)));
    gpuErrchk(hipMemcpy(gpuData, m.gpuData, d0 * d1 * sizeof(T), hipMemcpyDeviceToDevice));
}

template <class T>
CuMatrix<T>::~CuMatrix(void) {
    gpuErrchk(hipFree(gpuData));
}


template <class T>
void CuMatrix<T>::loadDataFrom(T *data) {
    // Malloc some GPU memory
    gpuErrchk(hipMalloc((void**)&gpuData, d0 * d1 * sizeof(T)));
    // Copy the data from the data buffer to the device
    gpuErrchk(hipMemcpy(gpuData, data, d0 * d1 * sizeof(T), hipMemcpyHostToDevice));
}

template <class T>
T* CuMatrix<T>::returnData() {
    T* data = new T[d0*d1];
    // Copy the data from the device to the data buffer
    gpuErrchk(hipMemcpy(data, gpuData, d0 * d1 * sizeof(T), hipMemcpyDeviceToHost));
    return data;
}

template <class T>
void CuMatrix<T>::transferData(T *newData) {
    gpuErrchk(hipFree(gpuData));
    gpuData = newData;
}

// Performs the operation C = A + B
template <class T>
void CuMatrix<T>::add(CuMatrix<T> &a, CuMatrix<T> &b, CuMatrix<T> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot add two dissimilar matrices";
    }
    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil((float)a.d0/dimBlock.x),(int)ceil((float)a.d1/dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixAdd<<<dimGrid, dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0 * a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

// Performs the operation C = A + vec * [1,1,...,1]
template <class T>
void CuMatrix<T>::addVector(CuMatrix<T> &a, CuMatrix<T> &vec, CuMatrix<T> &c) {
    if (a.d0 != vec.d0) {
        throw "Cannot add matrices with different number of rows";
    }
    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil((float)a.d0/dimBlock.x),(int)ceil((float)a.d1/dimBlock.y));
    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixAdd2<<<dimGrid, dimBlock>>>(a.gpuData, vec.gpuData, cData, a.d0 * a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

// Performs the operation C = A * B
template <>
void CuMatrix<float>::multiply(CuMatrix<float> &a, bool trA, CuMatrix<float> &b, bool trB, CuMatrix<float> &c) {
    if ((a.d0 != c.d0) || (b.d1 != c.d1) || (a.d1 != b.d0)) {
        throw "Matrix dimensions not correct";
    }

    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasOperation_t opA = trA? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = trB? HIPBLAS_OP_T : HIPBLAS_OP_N;
    // Do the actual multiplication
    hipblasSgemm(cuHandle, opA, opB, a.d0, b.d1, a.d1, alpha, a.gpuData, a.d0, b.gpuData, b.d0, beta, c.gpuData, c.d0);
}

// Performs the operation C = A x B where x is the Hadamard product
template <class T>
void CuMatrix<T>::hadm(CuMatrix<T> &a, CuMatrix<T> &b, CuMatrix<T> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot hadm two dissimilar matrices";
    }
    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil((float)a.d0/dimBlock.x),(int)ceil((float)a.d1/dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixHadm<<<dimGrid, dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0 * a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}
    
template <>
void CuMatrix<float>::applySigmoid() {
    dim3 dimBlock(32, 32);
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));
    matrixApplySigmoid<<<dimGrid, dimBlock>>>(gpuData, d0 * d1);
}

template <class T>
T CuMatrix<T>::reduce() {
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (d0 * d1 + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = nextpo2(blocksPerGrid);

    T *partial_sums = 0;
    gpuErrchk(hipMalloc((void**)&partial_sums, (blocksPerGrid + 1) * sizeof(T)));
    // Compute partial sums for all blocks
    reduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(T)>>>(gpuData, partial_sums, d0 * d1);
    // Launch a single block to compute sum of partial sums
    reduction<<<1, blocksPerGrid, blocksPerGrid * sizeof(T)>>>(partial_sums, partial_sums + blocksPerGrid, blocksPerGrid);

    T result = 0;
    gpuErrchk(hipMemcpy(&result, partial_sums + blocksPerGrid, sizeof(T), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(partial_sums));

    return result;
}