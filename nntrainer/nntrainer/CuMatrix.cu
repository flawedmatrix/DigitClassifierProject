#include "hip/hip_runtime.h"
#include "helpers.cuh"
#include "CuMatrix.cuh"
#include "kernels.cuh"
#include "hiprand.h"
#include <time.h>

hipblasHandle_t CuBase::cuHandle = NULL;

void CuBase::initializeHandle() {
    // Create a handle for CUBLAS
    hipblasCreate(&cuHandle);
}

void CuBase::closeHandle() {
    // Destroy the handle
    hipblasDestroy(cuHandle);
}

template <class T>
CuMatrixBase<T>::CuMatrixBase():
    d0(0), d1(0), gpuData(NULL), selection(NULL)
{
    dimBlock = dim3(32, 32);
}
template <class T>
CuMatrixBase<T>::CuMatrixBase(size_t rows, size_t cols):
    d0(rows), d1(cols), gpuData(NULL), selection(NULL)
{
    if (rows * 2 < cols) {
        dimBlock = dim3(4, 256);
    } else if (rows > cols * 2) {
        dimBlock = dim3(256, 4);
    } else {
        dimBlock = dim3(32, 32);
    }
}

template <class T>
CuMatrixBase<T>::CuMatrixBase(const CuMatrixBase<T> &m) {
    d0 = m.d0;
    d1 = m.d1;
    if (gpuData != NULL) {
        gpuErrchk(hipMalloc((void**)&gpuData, d0 * d1 * sizeof(T)));
        gpuErrchk(hipMemcpy(gpuData, m.gpuData, d0 * d1 * sizeof(T), hipMemcpyDeviceToDevice));
    }
    selection = NULL;
}

template <class T>
CuMatrixBase<T>::~CuMatrixBase(void) {
    gpuErrchk(hipFree(gpuData));
    gpuErrchk(hipFree(selection));
}
template <class T>
size_t CuMatrixBase<T>::getRows() {
    return d0;
}

template <class T>
size_t CuMatrixBase<T>::getCols() {
    return d1;
}

template <class T>
void CuMatrixBase<T>::loadDataFrom(T *data) {
    // Malloc some GPU memory
    gpuErrchk(hipMalloc((void**)&gpuData, d0 * d1 * sizeof(T)));
    // Copy the data from the data buffer to the device
    gpuErrchk(hipMemcpy(gpuData, data, d0 * d1 * sizeof(T), hipMemcpyHostToDevice));
}

template <class T>
void CuMatrixBase<T>::loadSelection(unsigned int *h_selection) { 
    gpuErrchk(hipMalloc((void**)&selection, d1 * sizeof(unsigned int)));
    gpuErrchk(hipMemcpy(selection, h_selection, d1 * sizeof(unsigned int), hipMemcpyHostToDevice));
}

template <class T>
void CuMatrixBase<T>::selectData(CuMatrixBase<T> &out, unsigned int blockStart, size_t n) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)n/dimBlock.y));

    T *tData;
    gpuErrchk(hipMalloc((void**)&tData, d0 * n * sizeof(T)));
    matrixSelectData<T><<<dimGrid, dimBlock>>>(gpuData, selection + blockStart, tData, d0, n);
    gpuErrchk(hipGetLastError());
    
    out.transferData(tData);
}

template <class T>
T* CuMatrixBase<T>::returnData() {
    T* data = new T[d0*d1];
    // Copy the data from the device to the data buffer
    gpuErrchk(hipMemcpy(data, gpuData, d0 * d1 * sizeof(T), hipMemcpyDeviceToHost));
    return data;
}

template <class T>
void CuMatrixBase<T>::transferData(T *newData) {
    gpuErrchk(hipFree(gpuData));
    gpuData = newData;
}

template <class T>
void CuMatrixBase<T>::fill(T num) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, d0 * d1 * sizeof(T)));
    matrixFill<T><<<dimGrid, dimBlock>>>(cData, num, d0, d1);
    gpuErrchk(hipGetLastError());
    transferData(cData);
}

template <class T>
void CuMatrixBase<T>::add(CuMatrixBase<T> &a, CuMatrixBase<T> &b, CuMatrixBase<T> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot add two dissimilar matrices";
    }
    dim3 dimGrid((int)ceil((float)a.d0/a.dimBlock.x),(int)ceil((float)a.d1/a.dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixAdd<T><<<dimGrid, a.dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0, a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

template <class T>
void CuMatrixBase<T>::sub(CuMatrixBase<T> &a, CuMatrixBase<T> &b, CuMatrixBase<T> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot sub two dissimilar matrices";
    }
    dim3 dimGrid((int)ceil((float)a.d0/a.dimBlock.x),(int)ceil((float)a.d1/a.dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixSub<T><<<dimGrid, a.dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0, a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

template <class T>
void CuMatrixBase<T>::addVector(CuMatrixBase<T> &a, CuMatrixBase<T> &vec, CuMatrixBase<T> &c) {
    if (a.d0 != vec.d0) {
        throw "Cannot add matrices with different number of rows";
    }
    dim3 dimGrid((int)ceil((float)a.d0/a.dimBlock.x),(int)ceil((float)a.d1/a.dimBlock.y));
    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixAdd2<T><<<dimGrid, a.dimBlock>>>(a.gpuData, vec.gpuData, cData, a.d0, a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

template <class T>
void CuMatrixBase<T>::hadm(CuMatrixBase<T> &a, CuMatrixBase<T> &b, CuMatrixBase<T> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot hadm two dissimilar matrices";
    }
    dim3 dimGrid((int)ceil((float)a.d0/a.dimBlock.x),(int)ceil((float)a.d1/a.dimBlock.y));

    T *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(T)));
    matrixHadm<T><<<dimGrid, a.dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0, a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

template <class T>
T CuMatrixBase<T>::reduce() {
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (d0 * d1 + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = nextpo2(blocksPerGrid);

    T *partial_sums = 0;
    gpuErrchk(hipMalloc((void**)&partial_sums, (blocksPerGrid + 1) * sizeof(T)));
    // Compute partial sums for all blocks
    reduction<T, T><<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(T)>>>(gpuData, partial_sums, d0 * d1);
    gpuErrchk(hipGetLastError());
    // Launch a single block to compute sum of partial sums
    reduction<T, T><<<1, blocksPerGrid, blocksPerGrid * sizeof(T)>>>(partial_sums, partial_sums + blocksPerGrid, blocksPerGrid);
    gpuErrchk(hipGetLastError());

    T result = 0;
    gpuErrchk(hipMemcpy(&result, partial_sums + blocksPerGrid, sizeof(T), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(partial_sums));

    return result;
}

int CuMatrix<char>::reduce() {
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (d0 * d1 + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = nextpo2(blocksPerGrid);

    int *partial_sums = 0;
    gpuErrchk(hipMalloc((void**)&partial_sums, (blocksPerGrid + 1) * sizeof(int)));
    // Compute partial sums for all blocks
    reduction<char, int><<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(gpuData, partial_sums, d0 * d1);
    gpuErrchk(hipGetLastError());
    // Launch a single block to compute sum of partial sums
    reduction<int, int><<<1, blocksPerGrid, blocksPerGrid * sizeof(int)>>>(partial_sums, partial_sums + blocksPerGrid, blocksPerGrid);
    gpuErrchk(hipGetLastError());

    int result = 0;
    gpuErrchk(hipMemcpy(&result, partial_sums + blocksPerGrid, sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(partial_sums));

    return result;
}

void CuMatrix<char>::encode(CuMatrix<float> &out, size_t d) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));

    float *tData;
    gpuErrchk(hipMalloc((void**)&tData, d * d0 * d1 * sizeof(float)));
    gpuErrchk(hipMemset(tData, 0, d * d0 * d1 * sizeof(float)));
    matrixEncode<<<dimGrid, dimBlock>>>(gpuData, tData, d, d0, d1);
    gpuErrchk(hipGetLastError());
    out.transferData(tData);
}

void CuMatrix<char>::notEquals(CuMatrix<char> &a, CuMatrix<char> &b, CuMatrix<char> &c) {
    if ((a.d0 != b.d0) || (a.d1 != b.d1)) {
        throw "Cannot xor two dissimilar matrices";
    }
    dim3 dimGrid((int)ceil((float)a.d0/a.dimBlock.x),(int)ceil((float)a.d1/a.dimBlock.y));

    char *cData;
    gpuErrchk(hipMalloc((void**)&cData, a.d0 * a.d1 * sizeof(char)));
    matrixNotEquals<<<dimGrid, a.dimBlock>>>(a.gpuData, b.gpuData, cData, a.d0, a.d1);
    gpuErrchk(hipGetLastError());
    c.transferData(cData);
}

void CuMatrix<char>::toFloat(CuMatrix<float> &target) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));

    float *tData;
    gpuErrchk(hipMalloc((void**)&tData, d0 * d1 * sizeof(float)));
    convertToFloat<<<dimGrid, dimBlock>>>(gpuData, tData, d0, d1);
    gpuErrchk(hipGetLastError());
    target.transferData(tData);
}

void CuMatrix<float>::multiply(CuMatrix<float> &a, bool trA, CuMatrix<float> &b, bool trB, CuMatrix<float> &c) {
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasOperation_t opA = trA? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = trB? HIPBLAS_OP_T : HIPBLAS_OP_N;
    unsigned int m = c.d0;
    unsigned int n = c.d1;
    unsigned int k = trA? a.d0 : a.d1;
    
    float *cData;
    gpuErrchk(hipMalloc((void**)&cData, m * n * sizeof(float)));
    // Do the actual multiplication
    CUBLAS_CALL(hipblasSgemm(cuHandle, opA, opB, m, n, k, alpha, a.gpuData, a.d0, b.gpuData, b.d0, beta, cData, c.d0));
    c.transferData(cData);
}

void CuMatrix<float>::applySigmoid() {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));
    matrixApplySigmoid<<<dimGrid, dimBlock>>>(gpuData, d0, d1);
    gpuErrchk(hipGetLastError());
}

void CuMatrix<float>::argmax(CuMatrix<char> &out) {
    // Spawn one thread per column of the matrix
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (d0 * d1 + threadsPerBlock - 1) / threadsPerBlock;

    char *tData;
    gpuErrchk(hipMalloc((void**)&tData, d1 * sizeof(int)));
    applyArgmax<float><<<blocksPerGrid, threadsPerBlock>>>(gpuData, tData, d0, d1);
    gpuErrchk(hipGetLastError());
    out.transferData(tData);
}

void CuMatrix<float>::scale(float factor) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));
    matrixScale<<<dimGrid, dimBlock>>>(gpuData, factor, d0, d1);
    gpuErrchk(hipGetLastError());
}

void CuMatrix<float>::normalize(float max) {
    dim3 dimGrid((int)ceil((float)d0/dimBlock.x),(int)ceil((float)d1/dimBlock.y));
    matrixNormalize<<<dimGrid, dimBlock>>>(gpuData, max, d0, d1);
    gpuErrchk(hipGetLastError());
}

void CuMatrix<float>::initRandom() {
    float *tData;
    gpuErrchk(hipMalloc((void**)&tData, d0 * d1 * sizeof(float)));
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    CURAND_CALL(hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set the seed for the random number generator using the system clock
    unsigned long long seed = (unsigned long long)clock();
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(prng, seed));
    // Fill the array with random numbers on the device
    CURAND_CALL(hiprandGenerateUniform(prng, tData, d0 * d1));
    transferData(tData);
    CURAND_CALL(hiprandDestroyGenerator(prng));
}

// Explicit declarations of the templated class
template class CuMatrixBase<char>;
template class CuMatrixBase<float>;